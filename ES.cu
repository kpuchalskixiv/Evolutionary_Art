#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hiprand.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <ES_kernel.h>


// utils



int cmp(const void *a, const void *b)
{
    struct loss_item *a1 = (struct loss_item *)a;
    struct loss_item *a2 = (struct loss_item *)b;
    if ((*a1).value < (*a2).value)
        return -1;
    else if ((*a1).value > (*a2).value)
        return 1;
    else
        return 0;
}
float sum_array(float* arr, int start, int end){
  int sum=0;
  for(int i=start; i<end; i++)
    sum+=arr[i];
  return sum;
}

void save_best(float* d_best_img, float* h_best_img,
              int img_x, int img_y, int iter){
  checkCudaErrors(hipMemcpy(h_best_img, d_best_img, //destination, source
                              img_x*img_y*sizeof(float),
                              hipMemcpyDeviceToHost) );
  //save to file
  FILE *fp;
  if(iter==1) fp = fopen("./cuda_output/mona_results.txt", "w");
  else fp = fopen("./cuda_output/mona_results.txt", "a");
  // check for error here

  fprintf(fp, "\n %d, ", iter);
  for (unsigned i = 0; i < img_x*img_y; i++) {
      fprintf(fp, "%f, ", h_best_img[i]);
      // check for error here too
  }

  fclose(fp);
}

void read_target(float* target_img,  int img_x, int img_y){
  FILE *fp;
  fp=fopen("./input/mona_gray_cuda.txt", "r");
  for(int i=0; i<img_x*img_y; i++){
    fscanf(fp, "%f", &target_img[i]);
  }
  fclose(fp);
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

 // bool gray=true;
  int genotype_length;
  int img_x=381, img_y=256;

  float *best_mate_img,
        *d_population,
        *d_population_copy,
        *d_population_images,
        *d_mutation_mates_coef, //*d_mutation_mates_if,
        *d_sigmas, *d_sigmas_copy,
        *d_mutation_sigmas_coef, *d_mutation_sigmas_if,
        *d_target_img, *h_target_img;
  

  //if(gray) genotype_length=6;
  //else
  genotype_length=8;

  size_t    bytes = (pop_size + children_per_mate*parents) *mate_size*genotype_length;
  size_t    eval_bytes =  (pop_size + children_per_mate*parents) *img_x*img_y;
  size_t    sigmas_bytes =  (pop_size + children_per_mate*parents)*genotype_length;

  // grid is popsizeXcircler_per_mateXcricledim (x1,y1,x2,y2,alpha,gray - cause black&white for start)
  printf("Grid dimensions: %d x %d x %d \n\n", pop_size, mate_size, genotype_length);
  // initialise card
  findCudaDevice(argc, argv);

  // allocate memory for arrays
  best_mate_img = (float *)malloc(sizeof(float) * img_y*img_x);
  h_target_img = (float *)malloc(sizeof(float) * img_y*img_x);

  read_target(h_target_img, img_x, img_y);
  printf("Target image read, sample pixel= %f\n", h_target_img[2137]);

  checkCudaErrors( hipMalloc((void **)&d_population, sizeof(float) * bytes) );
  checkCudaErrors( hipMalloc((void **)&d_mutation_mates_coef, sizeof(float) * bytes) );
  checkCudaErrors( hipMalloc((void **)&d_population_copy, sizeof(float) * bytes) );
  checkCudaErrors( hipMalloc((void **)&d_population_images, sizeof(float) * eval_bytes) );
  checkCudaErrors( hipMalloc((void **)&d_target_img, sizeof(float)  *img_x*img_y) );
  
  checkCudaErrors( hipMalloc((void **)&d_mutation_sigmas_coef, sizeof(float) *sigmas_bytes) );
  checkCudaErrors( hipMalloc((void **)&d_mutation_sigmas_if, sizeof(float) *sigmas_bytes) );
  checkCudaErrors( hipMalloc((void **)&d_sigmas, sizeof(float)* sigmas_bytes) );
  reset_values_kernel<<<pop_size+children_per_mate*parents, genotype_length>>>(d_sigmas, 0.5f, genotype_length);
  getLastCudaError("Reset kernel failed\n");
  checkCudaErrors( hipMalloc((void **)&d_sigmas_copy, sizeof(float) *sigmas_bytes) );


  for(int i=0; i<img_x*img_y; i++) h_target_img[i]=(float)rand()/(float)(RAND_MAX);
  checkCudaErrors(hipMemcpy(d_target_img, h_target_img, //destination, source
                              img_x*img_y*sizeof(float),
                              hipMemcpyHostToDevice) );

  // randomly initialize population
  
  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
 // checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateUniform(gen, d_population, bytes) );



  // Set up the execution configuration
  int iters=100000, log_every=1000,

  no_figures=1, update_frequency=5000;

  
  int last_update_iter=0;
  float *log_objective_values;
  float add_cricle_threshold=0.01, prev, curr;
  log_objective_values=(float *)malloc(sizeof(float)*iters);
  int iter;

  for (iter=0; iter<=iters; iter++) {
    if (iter%log_every==1){
      save_best(d_population_images+population_losses[0].index*img_x*img_y, 
                best_mate_img, 
                img_x, img_y, iter);
      printf("iter number: %d. Best MSE: %f \n", iter, population_losses[0].value);
 /*    printf("%f %d, %f %d, %f %d \n", 
      population_losses[0].value, population_losses[0].index,
      population_losses[pop_size-1].value, population_losses[pop_size-1].index,
      population_losses[pop_size+children_per_mate*parents-1].value, population_losses[pop_size+children_per_mate*parents-1].index
      );*/ 

    }
    
    // run selection, choose best parents and copy paste them into latter half of population array
    checkCudaErrors(hipMemcpy(d_population_copy, d_population, //destination, source
                            bytes*sizeof(float),
                            hipMemcpyDeviceToDevice) );
    checkCudaErrors(hipMemcpy(d_sigmas_copy, d_sigmas, //destination, source
                        sigmas_bytes*sizeof(float),
                        hipMemcpyDeviceToDevice) );
    hipDeviceSynchronize();  
    population_selection_kernel<<<parents, genotype_length*mate_size>>>(d_population, d_population_copy, genotype_length);
    getLastCudaError("Population selection kernel failed\n");
    population_selection_kernel<<<parents, genotype_length>>>(d_sigmas, d_sigmas_copy, genotype_length);
    getLastCudaError("Sigmas selection kernel failed\n");
    hipDeviceSynchronize();  
    checkCudaErrors(hipMemcpy(d_population, d_population_copy, //destination, source
                            bytes*sizeof(float),
                            hipMemcpyDeviceToDevice) );
    checkCudaErrors(hipMemcpy(d_sigmas, d_sigmas_copy, //destination, source
                            sigmas_bytes*sizeof(float),
                            hipMemcpyDeviceToDevice) );
    hipDeviceSynchronize();  

    //run mutation
    checkCudaErrors( hiprandGenerateNormal(gen, d_mutation_sigmas_coef, 
                                    sigmas_bytes,
                                    0.0f, 0.001f) );
    checkCudaErrors( hiprandGenerateUniform(gen, d_mutation_sigmas_if, sigmas_bytes));
    checkCudaErrors( hiprandGenerateUniform(gen, d_mutation_mates_coef, bytes));
    hipDeviceSynchronize();


    sigmas_mutation_kernel<<<1, sigmas_bytes>>>(d_sigmas,  d_mutation_sigmas_coef, d_mutation_sigmas_if);
    hipDeviceSynchronize();
    mate_mutation_kernel<<<pop_size+children_per_mate*parents, genotype_length*no_figures>>>(d_population, d_mutation_mates_coef, d_sigmas, genotype_length);
    getLastCudaError("Mutation kernel failed\n");
    hipDeviceSynchronize();    





    //run evaluation
    //mem set doesnt work with floats
   // checkCudaErrors(hipMemset(d_population_images, 0.0f, sizeof(float)*img_x*img_y*(pop_size+children_per_mate*parents)));

    reset_values_kernel<<<pop_size+children_per_mate*parents, 512>>>(d_population_images, 0.0f, img_x*img_y);
    getLastCudaError("Reset kernel failed\n");
    hipDeviceSynchronize();

    draw_kernel<<<pop_size+children_per_mate*parents, 512>>>(d_population, d_population_images, no_figures, img_x, img_y, genotype_length);
    getLastCudaError("Draw kernel failed\n");
    hipDeviceSynchronize();

    eval_kernel<<<pop_size+children_per_mate*parents, 512, 512*sizeof(float)>>>(d_population_images, 
                                   d_target_img, img_x, img_y);
    getLastCudaError("Eval kernel failed\n");
    hipDeviceSynchronize();

    qsort(population_losses, pop_size+children_per_mate*parents, sizeof(population_losses[0]), cmp);


    log_objective_values[iter]=population_losses[0].value;
    if(iter%update_frequency==0 
      && iter-last_update_iter>=update_frequency*2 
      && no_figures<=mate_size){
      prev=sum_array(log_objective_values, iter-2*update_frequency, iter-update_frequency);
      curr=sum_array(log_objective_values, iter-update_frequency, iter);

      if((prev-curr)/prev < add_cricle_threshold*powf(0.995, no_figures)){
        last_update_iter=iter;
        no_figures+=1;
        printf("Added rectangle at iter %d, current # is %d\n", iter, no_figures);
      }
      
    }
  }

 // Release GPU and CPU memory

  checkCudaErrors( hipFree(d_population) );
  checkCudaErrors( hipFree(d_population_images) );
  checkCudaErrors( hipFree(d_mutation_sigmas_coef) );

  free(best_mate_img);
  free(log_objective_values);

  hipDeviceReset();
}